#include "hip/hip_runtime.h"
#include <Eigen/Dense>

#include "../interface/inplace_fnnls.h"

using namespace Eigen;


// cache optimized matrix multiplication for 10x10 matrices
#ifdef __CUDA_ARCH__
__device__ __host__ 
#endif
inline FixedMatrix transpose_multiply(const FixedMatrix &A){
  FixedMatrix result;
  #pragma unroll MATRIX_SIZE
  for(auto i = 0; i < MATRIX_SIZE; ++i){    
    for(auto j = i; j < MATRIX_SIZE; ++j){
      result.data()[j*MATRIX_SIZE + i] = 0;
      #pragma vectorise
      for(auto k = 0; k < MATRIX_SIZE; ++k)
        result.data()[j*MATRIX_SIZE + i] += A.data()[i*MATRIX_SIZE+k]*A.data()[j*MATRIX_SIZE+k];
      result.data()[i*MATRIX_SIZE + j] = result.data()[j*MATRIX_SIZE + i];
    }
    // result = result.selfadjointView<Eigen::Upper>();
  }
  return result;
}

#ifdef __CUDA_ARCH__
__device__ __host__ 
#endif
void inplace_fnnls(const FixedMatrix& A,
                                       const FixedVector& b,
                                       FixedVector& x,
                                       const double eps,
                                       const unsigned int max_iterations) {
  // Fast NNLS (fnnls) algorithm as per
  // http://users.wfu.edu/plemmons/papers/Chennnonneg.pdf
  // page 8

  // FNNLS memorizes the A^T * A and A^T * b to reduce the computation.
  // The pseudo-inverse obtained has the same numerical problems so
  // I keep the same decomposition utilized for NNLS.

  // pseudoinverse (A^T * A)^-1 * A^T
  // this pseudo-inverse has numerical issues
  // in order to avoid that I substituted the pseudoinverse with the llT 
  // this require some modification the projection (A^T * A)^P
  // decomposition

  // NNLS initialization
  auto nPassive = 0;
  
  FixedMatrix AtA = transpose_multiply(A);
  FixedVector Atb = A.transpose() *b;

  FixedVector s;
  FixedVector w;

  Eigen::PermutationMatrix<VECTOR_SIZE> permutation;
  permutation.setIdentity();

// main loop
#pragma unroll VECTOR_SIZE
  for (auto iter = 0; iter < max_iterations; ++iter) {
    const auto nActive = VECTOR_SIZE - nPassive;

#ifdef DEBUG_FNNLS_CPU
    cout << "iter " << iter << endl;
#endif
    
    // If all the positivity contraints are satisfied the correct solution has 
    // been calculated, then exit.
    if(!nActive)
      break;

    w.tail(nActive) = Atb.tail(nActive) - (AtA * x).tail(nActive);

#ifdef DEBUG_FNNLS_CPU
    cout << "w" << endl << w.tail(nActive) << endl;
#endif
    // get the index of w that gives the maximum gain
    Index w_max_idx;
    const auto max_w = w.tail(nActive).maxCoeff(&w_max_idx);

    // check for convergence
    if (max_w < eps)
      break;
#ifdef DEBUG_FNNLS_CPU
    cout << "n active " << nActive << endl;
    cout << "w max idx " << w_max_idx << endl;
  #endif

    // need to translate the index into the right part of the vector
    w_max_idx += nPassive;

    // swap AtA to avoid copy
    AtA.col(nPassive).swap(AtA.col(w_max_idx));
    AtA.row(nPassive).swap(AtA.row(w_max_idx));
    // swap Atb to match with AtA
    Eigen::numext::swap(Atb.coeffRef(nPassive), Atb.coeffRef(w_max_idx));
    Eigen::numext::swap(x.coeffRef(nPassive), x.coeffRef(w_max_idx));
    // swap the permutation matrix to reorder the solution in the end
    Eigen::numext::swap(permutation.indices()[nPassive],
                        permutation.indices()[w_max_idx]);

    ++nPassive;

#ifdef DEBUG_FNNLS_CPU
    cout << "max index " << w_max_idx << endl;
    std::cout << "n_active " << nActive << std::endl;
#endif

// inner loop
#pragma unroll VECTOR_SIZE
    while (nPassive > 0) {
      // calculating the pseudoinverse ((A^T * A)^P)^-1 via llt decomposition
      s.head(nPassive) =
          AtA.topLeftCorner(nPassive, nPassive).llt().solve(Atb.head(nPassive));

      // If all the components are positive the solution has been calculated, 
      // then exit
      if (s.head(nPassive).minCoeff() > 0.) {
        x.head(nPassive) = s.head(nPassive);
        break;
      }

#ifdef DEBUG_FNNLS_CPU
      cout << "s" << endl << s.head(nPassive) << endl;
#endif
      // Compute the ratio to move a negative component to 0 
      auto alpha = std::numeric_limits<double>::max();
      Index alpha_idx = 0;

#pragma unroll VECTOR_SIZE
      for (auto i = 0; i < nPassive; ++i) {
        if (s[i] <= 0.) {
          auto const ratio = x[i] / (x[i] - s[i]);
          if (ratio < alpha) {
            alpha = ratio;
            alpha_idx = i;
          }
        }
      }
      // If no ratio can be derived all the components are not negative, exit
      if (std::numeric_limits<double>::max() == alpha) {
        x.head(nPassive) = s.head(nPassive);
        break;
      }

#ifdef DEBUG_FNNLS_CPU

      cout << "alpha " << alpha << endl;

      cout << "x before" << endl << x << endl;

#endif
      // translate the solution toward the positive part
      x.head(nPassive) += alpha * (s.head(nPassive) - x.head(nPassive));
      // This index will became 0 is set to this value to avoid numerical problems
      x[alpha_idx] = 0;
      --nPassive;

#ifdef DEBUG_FNNLS_CPU
      cout << "x after" << endl << x << endl;
#endif
      // new components sutisfy the non negativity contraints so is moved 
      // inside the active set
      AtA.col(nPassive).swap(AtA.col(alpha_idx));
      AtA.row(nPassive).swap(AtA.row(alpha_idx));
      // swap Atb to match with AtA
      Eigen::numext::swap(Atb.coeffRef(nPassive), Atb.coeffRef(alpha_idx));
      Eigen::numext::swap(x.coeffRef(nPassive), x.coeffRef(alpha_idx));
      // swap the permutation matrix to reorder the solution in the end
      Eigen::numext::swap(permutation.indices()[nPassive],
                          permutation.indices()[alpha_idx]);
    }
  }
  // reshuffle the solution to the original order
  x = x.transpose() * permutation.transpose();  
}


/*
#ifdef __CUDA_ARCH__
__global__
#endif
void matrixMultiplicationKernel(const double* A, double* C);
// #endif // __CUDA_ARCH__


#ifdef __CUDA_ARCH__
__global__ void matrixMultiplicationKernel(const double* A, double* C) {

  int x = blockIdx.y*blockDim.y+threadIdx.y;
  int y = blockIdx.x*blockDim.x+threadIdx.x;
  // printf("x %d y %d\n",x, y);
  // printf("%f ", C[y * MATRIX_SIZE + x] );
  C[y * MATRIX_SIZE + x] = 0.;
  // printf("%f ", C[y * MATRIX_SIZE + x] );
  // for (int i = 0; i < MATRIX_SIZE; i++) {
    // printf("B[%d][%d]= \n",y,x);
    // C[y * MATRIX_SIZE + x] += 
        // A[y * MATRIX_SIZE + i] * A[x * MATRIX_SIZE + i];
        // printf("A[%d][%d]*A[%d][%d]\n",y, i, x, i);
      // }
  // __syncthreads();
  // C->data()[ROW * MATRIX_SIZE + y] = C->data()[y * MATRIX_SIZE + ROW];
}

__device__
void matrixMultiplication(const FixedMatrix &A, FixedMatrix &result){
  // declare the number of blocks per grid and the number of threads per block
  // use 1 to 512 threads per block
  // dim3 threadsPerBlock(1, 1);
  dim3 threadsPerBlock(MATRIX_SIZE, MATRIX_SIZE);
  dim3 blocksPerGrid(1, 1);
  // for(auto i = 0; i < MATRIX_SIZE; ++i){
    // for(auto j = 0; j < MATRIX_SIZE; ++j)
      // result(i,j) = 0.;
  // }
  matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A.data(), result.data());
}

#endif

*/